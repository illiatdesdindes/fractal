#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   cudafractal.cu                                     :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: svachere <svachere@student.42.fr>          +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2015/01/28 13:53:04 by svachere          #+#    #+#             */
/*   Updated: 2015/01/28 13:53:04 by svachere         ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include <fractol.h>

int		julia(t_env *env, int x, int y, int max)
{
	float		temp_x;
	float		temp_y;
	float		coord_x;
	float		coord_y;
	int			i;

	coord_x = (((x + env->xzoom) / env->zoom / WIN_WIDTH) * 4 - 2);
	coord_y = (((y + env->yzoom) / env->zoom / WIN_HEIGHT) * 4 - 2);
	i = 0;
	while (((coord_x * coord_x) + (coord_y * coord_y)) < 4 && i < max)
	{
		temp_x = coord_x;
		temp_y = coord_y;
		coord_x = (temp_x * temp_x) - (temp_y * temp_y) + env->dx;
		coord_y = (temp_x * temp_y) + (temp_x * temp_y) + env->dy;
		i++;
	}
	return (i);
}

int		mandelbrot(t_env *env, int x, int y, int max)
{
	float		temp_x;
	float		temp_y;
	float		coord_x;
	float		coord_y;
	int			i;

	coord_x = 0.0;
	coord_y = 0.0;
	i = 0;
	while (((coord_x * coord_x) + (coord_y * coord_y)) < 4 && i < max)
	{
		temp_x = coord_x;
		temp_y = coord_y;
		coord_x = (temp_x * temp_x) - (temp_y * temp_y) + \
				(((x + env->xzoom) / env->zoom / WIN_WIDTH) * 4 - 2);
		coord_y = (temp_x * temp_y) + (temp_x * temp_y) + \
				(((y + env->yzoom) / env->zoom / WIN_HEIGHT) * 4 - 2);
		i++;
	}
	return (i);
}

int		sierpinski(t_env *env, int x, int y, int max)
{
	int		i;
	int		temp;

	x = ((x + env->xzoom) / env->zoom);
	if (x < 0)
		x = -x;
	y = ((y + env->yzoom) / env->zoom);
	if (y < 0)
		y = -y;
	temp = max;
	i = 0;
	while ((x > 0 || y > 0) && i < temp)
	{
		if ((x % 3) == 1 && (y % 3) == 1)
			return (i);
		x /= 3;
		y /= 3;
		i++;
	}
	return (temp);
}



void	cuda_draw_frac(t_env *env, int max)
{
	while (x <= WIN_WIDTH)
	{
		y = 0
		while (y <= WIN_HEIGHT)
		{
			if (env->choice == 's')
				put_pxl_img(env, x, y, get_clr(sierpinski(env, x, y, max), \
							max, env->clr));
			else if (env->choice == 'm')
				put_pxl_img(env, x, y, get_clr(mandelbrot(env, x, y, max), \
							max, env->clr));
			else if (env->choice == 'j')
				put_pxl_img(env, x, y, get_clr(julia(env, x, y, max), \
							max, env->clr));
			y++;
		}
		x++;
	}
}
